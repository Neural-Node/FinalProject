
#include <hip/hip_runtime.h>
__device__ void mergeGPU(int *arr, int *temp, int l, int m, int r, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int i = l + tid;
    int j = m + 1 + tid;
    int k = l + tid;

    while (i <= m && j <= r && k < n) {
        if (arr[i] <= arr[j])
            temp[k - l] = arr[i++];
        else
            temp[k - l] = arr[j++];
        k++;
    }

    while (i <= m && k < n)
        temp[k++ - l] = arr[i++];

    while (j <= r && k < n)
        temp[k++ - l] = arr[j++];

    __syncthreads();

    for (i = l + tid; i <= r && i < n; i += blockDim.x * gridDim.x)
        arr[i] = temp[i - l];
}

__global__ void mergeSort_GPU(int *arr, int *temp, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int l, r, m;

    for (int size = 1; size < n; size *= 2) {
        for (int start = 0; start < n - 1; start += 2 * size) {
            l = start;
            m = start + size - 1;
            r = min(start + 2 * size - 1, n - 1);
            mergeGPU(arr, temp, l, m, r, n);
        }
        __syncthreads();
    }
}
