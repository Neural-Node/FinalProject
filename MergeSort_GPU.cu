#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256

__device__ void mergeGPU(int *arr, int *temp, int l, int m, int r) {
    int i = l + threadIdx.x;
    int j = m + 1 + threadIdx.x;
    int k = l + threadIdx.x;
    
    // Merge the two sorted arrays into temp array
    while (i <= m && j <= r) {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }
    
    // Copy the remaining elements of left array, if any
    while (i <= m)
        temp[k++] = arr[i++];
    
    // Copy the remaining elements of right array, if any
    while (j <= r)
        temp[k++] = arr[j++];
    
    // Copy the merged portion back to the original array
    for (i = l + threadIdx.x; i <= r; i += blockDim.x)
        arr[i] = temp[i];
}

__global__ void mergeSort_GPU_kernel(int *arr, int *temp, int n) {
    // Calculate thread and block indices
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Merge sort algorithm
    for (int size = 1; size < n; size *= 2) {
        for (int start = tid; start < n - 1; start += stride * size) {
            int l = start;
            int m = min(start + size - 1, n -1);
            int r = min(start + 2 * size - 1, n - 1);
            mergeGPU(arr, temp, l, m, r);    
        }
	__syncthreads(); //Ensure all theads completed before proceeding
    }
    
}

void mergeSort_GPU(int *arr, int *temp, int n, int *sorted_arr) {
    int *arr_gpu, *temp_gpu;
    hipMalloc((void**)&arr_gpu, n * sizeof(int));
    hipMalloc((void**)&temp_gpu, n * sizeof(int));

    hipMemcpy(arr_gpu, arr, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    mergeSort_GPU_kernel<<<blocks, THREADS_PER_BLOCK>>>(arr_gpu, temp_gpu, n);

    hipMemcpy(sorted_arr, arr_gpu, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(arr_gpu);
    hipFree(temp_gpu);
}