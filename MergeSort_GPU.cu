#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256

__device__ void mergeGPU(int *arr, int *temp, int l, int m, int r) {
    int i = l + threadIdx.x;
    int j = m + 1 + threadIdx.x;
    int k = l + threadIdx.x;
    
    // Merge the two sorted arrays into temp array
    while (i <= m && j <= r) {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }
    
    // Copy the remaining elements of left array, if any
    while (i <= m)
        temp[k++] = arr[i++];
    
    // Copy the remaining elements of right array, if any
    while (j <= r)
        temp[k++] = arr[j++];
    
    // Copy the merged portion back to the original array
    for (i = l + threadIdx.x; i <= r; i += blockDim.x)
        arr[i] = temp[i];
}

__global__ void mergeSort_GPU(int *arr, int *temp, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int l, r, m;
    
    // Merge sort algorithm
    for (int size = 1; size < n; size *= 2) {
        for (int start = 0; start < n - 1; start += 2 * size) {
            l = start;
            m = start + size - 1;
            r = min(start + 2 * size - 1, n - 1);
            mergeGPU(arr, temp, l, m, r);
        }
    }
}