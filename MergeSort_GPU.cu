#include "hip/hip_runtime.h"
#include "MergeSort_GPU.h"
#include <stdio.h>

#define THREADS_PER_BLOCK 256

__global__ void merge(int *arr, int *temp, int l, int m, int r) {
    int i = l + blockIdx.x * blockDim.x + threadIdx.x;
    int j = m + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    int k = l + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Merge the two sorted arrays into temp array
    while (i <= m && j <= r) {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }
    
    // Copy the remaining elements of left array, if any
    while (i <= m)
        temp[k++] = arr[i++];
    
    // Copy the remaining elements of right array, if any
    while (j <= r)
        temp[k++] = arr[j++];
    
    // Copy the merged portion back to the original array
    for (i = l + blockIdx.x * blockDim.x + threadIdx.x; i <= r; i += blockDim.x * gridDim.x)
        arr[i] = temp[i];
}

__global__ void mergeSort(int *arr, int *temp, int l, int r) {
    if (l < r) {
        int m = l + (r - l) / 2;
        
        // Recursively sort the two halves
        mergeSort<<<(r - l + 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(arr, temp, l, m);
        mergeSort<<<(r - l + 1) / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(arr, temp, m + 1, r);
        
        // Merge the sorted halves
        merge<<<1, THREADS_PER_BLOCK>>>(arr, temp, l, m, r);
    }

}
