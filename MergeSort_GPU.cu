#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 1024

__device__ void Merge(int* arr, int* temp, int left, int middle, int right) {
    int i = left;
    int j = middle;
    int k = left;

    while (i < middle && j < right) {
        if (arr[i] <= arr[j])
            temp[k++] = arr[i++];
        else
            temp[k++] = arr[j++];
    }

    while (i < middle)
        temp[k++] = arr[i++];
    while (j < right)
        temp[k++] = arr[j++];

    for (int x = left; x < right; x++)
        arr[x] = temp[x];
}

__global__ void MergeSortGPU(int* arr, int* temp, int n, int width) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int left = tid * width;
    int middle = left + width / 2;
    int right = left + width;

    if (left < n && middle < n) {
        Merge(arr, temp, left, middle, right);
    }
}

void mergeSort_GPU(int *arr, int n, int *sorted_arr) {
    int *arr_gpu, *temp_gpu;
    hipMalloc((void**)&arr_gpu, n * sizeof(int));
    hipMalloc((void**)&temp_gpu, n * sizeof(int));

    hipMemcpy(arr_gpu, arr, n * sizeof(int), hipMemcpyHostToDevice);

    int blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int width = 2; // initial width for merging
    
    while (width < n) {
        MergeSortGPU<<<blocks, THREADS_PER_BLOCK>>>(arr_gpu, temp_gpu, n, width);
        width *= 2; // double the width for next iteration
    }

    hipMemcpy(sorted_arr, arr_gpu, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(arr_gpu);
    hipFree(temp_gpu);
}