#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "MergeSort_CPU.h"

// Define your GPU merge sort function here
void mergeSort_GPU(int *arr, int n, int *sorted_arr);

#define MAX_SIZE 1000

int main() {
    int n;

    // Input size of the array
    printf("Enter the size of the array: ");
    scanf("%d", &n);

    if (n <= 0 || n > MAX_SIZE) {
        printf("Invalid size.\n");
        return 1;
    }

    int *arr = (int *)malloc(n * sizeof(int));
    int *sorted_arr = (int *)malloc(n * sizeof(int));

    // Generate random array
    srand(time(NULL));
    printf("\nOriginal Array:\n");
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 100; // Generate random values between 0 and 99
        printf("%d ", arr[i]);
    }
    printf("\n");

    // Menu for sorting options
    int choice;
    printf("\nChoose sorting option: \n");
    printf("1. CPU Merge Sort\n");
    printf("2. GPU Merge Sort \n");
    printf("Enter your choice: ");
    scanf("%d", &choice);

    switch (choice) {
        case 1: {
            mergeSort_CPU(arr, 0, n - 1, sorted_arr);
            printf("\nSorted Array using CPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", sorted_arr[i]); // Print the sorted array
            }
            printf("\n");
            break;
        }
        case 2: {
            // Call GPU merge sort
            mergeSort_GPU(arr, n, sorted_arr);
            printf("\nSorted Array using GPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", sorted_arr[i]); // Print the sorted array
            }
            printf("\n");
            break;
        }
        default:
            printf("Invalid choice\n");
            break;
    }

    // Free dynamically allocated memory
    free(arr);
    free(sorted_arr);

    return 0;
}
