#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "MergeSort_CPU.h"
#include "MergeSort_GPU.h"

#define MAX_SIZE 1000
#define THREADS_PER_BLOCK 256

int main() {
    int n;

    // Input size of the array
    printf("Enter the size of the array: ");
    scanf("%d", &n);

    if (n <= 0 || n > MAX_SIZE) {
        printf("Invalid size.\n");
        return 1;
    }

    int *arr = (int *)malloc(n * sizeof(int));
    int *arr_copy = (int *)malloc(n * sizeof(int));

    // Generate random array
    srand(time(NULL));
    printf("\nOriginal Array:\n");
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 100; // Generate random values between 0 and 99
        arr_copy[i] = arr[i];  // Make a copy for GPU merge sort
        printf("%d ", arr[i]);
    }
    printf("\n");

    // Menu for sorting options
    int choice;
    printf("\nChoose sorting option: \n");
    printf("1. CPU Merge Sort\n");
    printf("2. GPU Merge Sort \n");
    printf("Enter your choice: ");
    scanf("%d", &choice);

    int *temp_gpu = NULL; // Initialize temp_gpu pointer outside the switch

    switch (choice) {
        case 1:
            mergeSort_CPU(arr, 0, n - 1); // Sort the array using CPU merge sort
            printf("\nSorted Array using CPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", arr[i]);
            }
            break;

        case 2:
            temp_gpu = (int *)malloc(n * sizeof(int)); // Allocate memory for temporary array
            mergeSort_GPU<<<1, THREADS_PER_BLOCK>>>(arr_copy, temp_gpu, n); // Launch GPU merge sort
            hipDeviceSynchronize(); // Wait for GPU to finish
            printf("\nSorted Array using GPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", arr_copy[i]);
            }
            printf("\n");
            free(temp_gpu); // Free temporary array memory
            break;

        default:
            printf("Invalid choice\n");
            break;
    }

    // Free dynamically allocated memory
    free(arr);
    free(arr_copy);

    return 0;
}
