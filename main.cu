#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "MergeSort_CPU.h"

// Define your GPU merge sort function here
void mergeSort_GPU(int *arr, int n, int *sorted_arr);

#define MAX_SIZE 1000000000000

int main() {
    int n;

    // Input size of the array
    printf("Enter the size of the array: ");
    scanf("%d", &n);

    if (n <= 0 || n > MAX_SIZE) {
        printf("Invalid size.\n");
        return 1;
    }

    int *arr_cpu = (int *)malloc(n * sizeof(int));
    int *arr_gpu = (int *)malloc(n * sizeof(int));
    int *sorted_arr_cpu = (int *)malloc(n * sizeof(int));
    int *sorted_arr_gpu = (int *)malloc(n * sizeof(int));

    // Generate random arrays
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        arr_cpu[i] = rand() % 100; // Generate random values between 0 and 99
        arr_gpu[i] = arr_cpu[i];
    }

    // CPU Merge Sort
    clock_t start_cpu, end_cpu;
    double cpu_time_used;

    start_cpu = clock();
    mergeSort_CPU(arr_cpu, 0, n - 1, sorted_arr_cpu);
    end_cpu = clock();
    cpu_time_used = ((double) (end_cpu - start_cpu)) / CLOCKS_PER_SEC;
    printf("\nSorted Array using CPU Merge Sort:\n");
    printf("Time elapsed for CPU Merge Sort: %f seconds\n", cpu_time_used);

    // GPU Merge Sort
    clock_t start_gpu, end_gpu;
    double gpu_time_used;

    start_gpu = clock();
    mergeSort_GPU(arr_gpu, n, sorted_arr_gpu);
    end_gpu = clock();
    gpu_time_used = ((double) (end_gpu - start_gpu)) / CLOCKS_PER_SEC;
    printf("\nSorted Array using GPU Merge Sort:\n");
    printf("Time elapsed for GPU Merge Sort: %f seconds\n", gpu_time_used);

    // Free dynamically allocated memory
    free(arr_cpu);
    free(arr_gpu);
    free(sorted_arr_cpu);
    free(sorted_arr_gpu);

    return 0;
}
