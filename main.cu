#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "MergeSort_CPU.h"
#include "MergeSort_GPU.h"

#define MAX_SIZE 1000

int main() {
    int n;

    // Input size of the array
    printf("Enter the size of the array: ");
    scanf("%d", &n);

    if (n <= 0 || n > MAX_SIZE) {
        printf("Invalid size.\n");
        return 1;
    }

    int *arr = (int *)malloc(n * sizeof(int));
    int *arr_copy = (int *)malloc(n * sizeof(int));
    int *sorted_arr = (int *)malloc(n * sizeof(int));

    // Generate random array
    srand(time(NULL));
    printf("\nOriginal Array:\n");
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 100; // Generate random values between 0 and 99
        arr_copy[i] = arr[i];  // Make a copy for GPU merge sort
        printf("%d ", arr[i]);
    }
    printf("\n");

    // Menu for sorting options
    int choice;
    printf("\nChoose sorting option: \n");
    printf("1. CPU Merge Sort\n");
    printf("2. GPU Merge Sort \n");
    printf("Enter your choice: ");
    scanf("%d", &choice);


    switch (choice) {
        case 1: {
	    // Timing CPU mergeSort
	    hipEvent_t start, end;
	    hipEventCreate(&start);
	    hipEventCreate(&end);
            hipEventRecord(start);

            mergeSort_CPU(arr, 0, n - 1,sorted_arr);

            hipEventRecord(end);
            hipEventSynchronize(end);
            float elapsed_time;
            hipEventElapsedTime(&elapsed_time, start, end);
            printf("\nSorted Array using CPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", sorted_arr[i]); // Print the sorted array
            }
	    printf("\nTime elapsed %.6f ms\n", elapsed_time);
            hipEventDestroy(start);
	    hipEventDestroy(end);
            break;
        }
        case 2:
	    // Timing GPU mergeSort
            hipEvent_t start, end;
            hipEventCreate(&start);
	    hipEventCreate(&end);
            hipEventRecord(start);

            mergeSort_GPU(arr_copy,0, n, sorted_arr);

	    hipEventRecord(end);
	    hipEventSynchronize(end);
            float elapsed_time;
	    hipEventElapsedTime(&elapsed_time, start, end);
            printf("\nSorted Array using GPU Merge Sort:\n");
            for (int i = 0; i < n; i++) {
                printf("%d ", sorted_arr[i]);
            }
	    printf("\nTime elapsed: %.6f ms\n", elapsed_time);
	    hipEventDestroy(start);
	    hipEventDestroy(end);
            break;

        default:
            printf("Invalid choice\n");
            break;
    }

    // Free dynamically allocated memory
    free(arr);
    free(arr_copy);
    free(sorted_arr);

    return 0;
}

